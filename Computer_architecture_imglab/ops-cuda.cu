#include "hip/hip_runtime.h"
/*
 * Skeleton code for use with Computer Architecture 2018 assignment 3,
 * LIACS, Leiden University.
 * 
 * From Dennis Buurman (s2027100) and Jerry Schonenberg (s2041022)
 */

#include "run.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

/* Some simple assert macro and inline function to handle CUDA-errors
 * more easily.
 */
#define CUDA_ASSERT(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void
cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
              file, line);

      if (abort)
        exit(code);
    }
}

/*
 * Forward defines for the 2D lighting kernel.
 */

typedef struct
{
  // Location in relative coordinates (0,0) = top left, (1,1) = bottom right
  float x, y;
  // Height of the light above the image plane, for faux 3D effect
  float height;
  // Intensity and color of the light, alpha value is unused
  rgba_t intensity;
} light_t;

/* TODO: enable once you need these */
#if 0
/* We add a certain constant value to z to smooth out shading, 1.1
 * was empirically determined to work well :)
 */
static const float SMOOTH_OUT_FACTOR = 1.1f;

/* This constant is used to scale the heightmap values (0.0-1.0) to the
 * image domain. Note that increasing this values reduces round-off
 * errors in exchange for worse performance.
 */
static const float HEIGHT_SCALE = 128.0f;

/* The color of the ambient light. */
static const rgba_t AMBIENT = { 0.1, 0.1, 0.2, 1.0 };
#endif


/*
 * CUDA kernels
 */

/* Example kernel for an image copy operation. */
__global__ void
op_cuda_copy(uint32_t *dst, const uint32_t *src, const int rowstride,
             const int x, const int y,
             const int width, const int height)
{
  const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx < x || xx >= width || yy < y || y >= height)
    return;

  /* Get the pixel in src and store in dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, xx, yy) = pixel;
}

__global__ void
op_cuda_transpose(uint32_t *dst, const uint32_t *src, const int rowstride,
             			const int x, const int y,
             			const int width, const int height)
{	
	const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (xx < x || xx >= width || yy < y || y >= height)
    return;
  
  /* Get the pixel in src and transpose into dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, yy, xx) = pixel;
}

/* Computes the intensity of @color. To do so, we use CIE 1931 weights
 * multiplied by alpha: Y = A( 0.2126R + 0.7152G + 0.0722B ).
 */
__device__ static float
compute_intensity(rgba_t color)
{
	return color.w * (0.2126f * color.x + 0.7152f * color.y + 0.0722 * color.z);
}

/* Computes the image histogram of @image, based on grayscale intensity.
 * The histogram is saved in the @bins argument.
 */
__global__ void //TODO vragen: - 3 pixels verschillen met origineel
op_cuda_histogram(unsigned int bins_D[], const uint32_t *src, const int rowstride,
             			const int x, const int y,
             			const int width, const int height)
{
	const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (xx < x || xx >= width || yy < y || y >= height)
    return;
  
  rgba_t color;
  RGBA_unpack(color, *image_get_pixel_data(src, rowstride, xx, yy));
  
  unsigned int idx = (unsigned int)(compute_intensity(color) * N_BINS);
  idx = MIN(N_BINS-1, idx);
  atomicAdd(&bins_D[idx], 1);
}

__device__ static int
compute_hue (rgba_t color)
{
	float hue = 0.f;

  /* Find the 'largest' component of either of r, g or b */
  float c_max = fmax(fmax(color.x, color.y), color.z);
  float c_min = fmin(fmin(color.x, color.y), color.z);
	
  /* The hue depends on which component in the largest */
  if (c_max == color.x)
    hue = (color.y - color.z) / (c_max - c_min);
  else if (c_max == color.y)
    hue = 2.f + (color.z - color.x) / (c_max - c_min);
  else
    hue = 4.f + (color.x - color.y) / (c_max - c_min);
	
  /* Ensure hue is in the range of [0-360]. */
  hue *= 60.f;
  if (hue < 0)
    hue += 360.f;  
  return (int)hue;
}

__global__ void
op_cuda_selgray(uint32_t *src, const int rowstride, const int x, const int y,
								const int width, const int height, const int hue, 
								const int spread)
{
	const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (xx < x || xx >= width || yy < y || y >= height)
    return;
  
  rgba_t color;
  RGBA_unpack(color, *image_get_pixel_data(src, rowstride, xx, yy));

  /* Compute the 'hue' (tint) and the difference with the
   * given hue...
   */
  int diff = (int)abs(hue - compute_hue(color));

  /* ...this difference determines whether we pick the gray or the
   * original color. We use a linear weight to reduce artifacts in
   * the final image.
   */
  float weight = (diff <= spread) ? (float)diff / (float)spread : 1.f;

	/* Apply the weights to the 'color' and 'gray' components. */
  rgba_t gray;
  float intensity = compute_intensity(color);
  
  RGBA(gray, intensity, intensity, intensity, 1.f);

  RGBA_mults(gray, gray, weight);
  
  RGBA_mults(color, color, 1.f-weight);
  
  /* Finally, add both components to produce the resulting pixel. */
  RGBA_add(color, color, gray);    
            
  RGBA_pack(*image_get_pixel_data(src, rowstride, xx, yy), color);
}

extern "C"
{

image_t *
run_op_transpose(const image_t *source, const int n_repeat)
{
	/* Allocate buffers to contain initial image and result */
	uint32_t *result_D; uint32_t *image_D;
	int n_pixels = source->height * source->width;
	int n_bytes = n_pixels * sizeof(uint32_t);
	
	CUDA_ASSERT(hipMalloc(&result_D, n_bytes));
	CUDA_ASSERT(hipMalloc(&image_D, n_bytes));
	
  /* Calculate the block size and the number of thread blocks */
  const dim3 blocksize(16, 16);
  const dim3 numblocks((source->width % blocksize.x) ?
                          source->width / blocksize.x + 1 :
                          source->width / blocksize.x,
                       (source->height % blocksize.y) ?
                          source->height / blocksize.y + 1 :
                          source->height / blocksize.y);

	/* Copy the input image to the initial buffer */
	CUDA_ASSERT(hipMemcpy(image_D, source->data, n_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  CUDA_ASSERT(hipEventCreate(&start));
  CUDA_ASSERT(hipEventCreate(&stop));

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  for (int Z = 0; Z < n_repeat; Z++)
    {
      op_cuda_transpose<<<numblocks, blocksize>>>(result_D, image_D,
                                                  source->rowstride,
                                                  0, 0,
                                                  source->width,
                                                  source->height);
      CUDA_ASSERT(hipGetLastError());
    }

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  image_t *result = image_new_from_image_T(source);
  
	CUDA_ASSERT(hipMemcpy(result->data, result_D, n_bytes, hipMemcpyDeviceToHost));

  float msec = 0;
  
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));
  
  CUDA_ASSERT(hipFree(result_D));
  CUDA_ASSERT(hipFree(image_D));
  
  fprintf(stderr, "elapsed time GPU transpose: %f s\n", msec / 1000.);

  return result;
}


void
run_op_histogram(const image_t *source, const int n_repeat)
{
  /* Allocate buffers to contain initial image and result */
  uint32_t *image_D;
  unsigned int *bins_D;
  unsigned int bins[N_BINS];
	int n_pixels = source->height * source->width;
	int n_bytes = n_pixels * sizeof(uint32_t);
	
	for (int i = 0; i < N_BINS; i++){
		bins[i] = 0;
	}
	
	CUDA_ASSERT(hipMalloc(&image_D, n_bytes));
	CUDA_ASSERT(hipMalloc(&bins_D, N_BINS*sizeof(unsigned int)));
	
  /* Calculate the block size and the number of thread blocks */
	const dim3 blocksize(16, 16);
  const dim3 numblocks((source->width % blocksize.x) ?
                          source->width / blocksize.x + 1 :
                          source->width / blocksize.x,
                       (source->height % blocksize.y) ?
                          source->height / blocksize.y + 1 :
                          source->height / blocksize.y);
                       
	/* Copy the input image to the initial buffer */
	CUDA_ASSERT(hipMemcpy(image_D, source->data, n_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  for (int Z = 0; Z < n_repeat; Z++)
    {
      CUDA_ASSERT(hipMemcpy(bins_D, bins, N_BINS*sizeof(unsigned int), 
      						hipMemcpyHostToDevice));
      op_cuda_histogram<<<numblocks, blocksize>>>(bins_D, image_D,
                                                  source->rowstride,
                                                  0, 0,
                                                  source->width,
                                                  source->height);
      CUDA_ASSERT(hipGetLastError());
    }

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

	CUDA_ASSERT(hipMemcpy(bins, bins_D, N_BINS*sizeof(unsigned int), hipMemcpyDeviceToHost));

  float msec = 0;
  
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));
  fprintf(stderr, "elapsed time GPU histogram: %f s\n", msec / 1000.);

  CUDA_ASSERT(hipFree(image_D));
  CUDA_ASSERT(hipFree(bins_D));
  
  /* Print result to stdout. */
  for (int i = 0; i < N_BINS; i++)
    printf("%d ", bins[i]);
  printf("\n");
}

image_t *
run_op_transgram(const image_t *source, const int n_repeat)
{
  /* Allocate buffers to contain initial image and result */
	uint32_t *image_D; uint32_t *result_D;
  unsigned int *bins_D;
  unsigned int bins[N_BINS];
	int n_pixels = source->height * source->width;
	int n_bytes = n_pixels * sizeof(uint32_t);
	
	for (int i = 0; i < N_BINS; i++){
		bins[i] = 0;
	}
	
  /*Copy the input image to the initial buffer */
	CUDA_ASSERT(hipMalloc(&image_D, n_bytes));
	CUDA_ASSERT(hipMalloc(&bins_D, N_BINS*sizeof(unsigned int)));
	CUDA_ASSERT(hipMalloc(&result_D, n_bytes));
  
  /*Calculate the block size and the number of thread blocks */
	const dim3 blocksize(16, 16);
  const dim3 numblocks((source->width % blocksize.x) ?
                          source->width / blocksize.x + 1 :
                          source->width / blocksize.x,
                       (source->height % blocksize.y) ?
                          source->height / blocksize.y + 1 :
                          source->height / blocksize.y);

	/* Copy the input image to the initial buffer */
	CUDA_ASSERT(hipMemcpy(image_D, source->data, n_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  for (int Z = 0; Z < n_repeat; Z++)
    {
      CUDA_ASSERT(hipMemcpy(bins_D, bins, N_BINS*sizeof(unsigned int), 
      						hipMemcpyHostToDevice));
      
      op_cuda_transpose<<<numblocks, blocksize>>>(result_D, image_D,
                                                  source->rowstride,
                                                  0, 0,
                                                  source->width,
                                                  source->height);
      
      op_cuda_histogram<<<numblocks, blocksize>>>(bins_D, result_D,
                                                  source->rowstride,
                                                  0, 0,
                                                  source->width,
                                                  source->height);
      
      CUDA_ASSERT(hipGetLastError());
    }

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  image_t *result = image_new_from_image_T(source);
  
  /*copy results back */
	CUDA_ASSERT(hipMemcpy(bins, bins_D, N_BINS*sizeof(unsigned int), hipMemcpyDeviceToHost));
	CUDA_ASSERT(hipMemcpy(result->data, result_D, n_bytes, hipMemcpyDeviceToHost));

  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));
  fprintf(stderr, "elapsed time GPU transgram: %f s\n", msec / 1000.);

	CUDA_ASSERT(hipFree(image_D));
  CUDA_ASSERT(hipFree(bins_D));
  CUDA_ASSERT(hipFree(result_D));

  /* Print result to stdout. */
  for (int i = 0; i < N_BINS; i++)
    printf("%d ", bins[i]);
  printf("\n");

  return result;
}

void
run_op_selgray(image_t *source, const int n_repeat)
{
  /*Allocate buffers to contain image */
	uint32_t *image_D;
	int n_pixels = source->height * source->width;
	int n_bytes = n_pixels * sizeof(uint32_t);
	
	CUDA_ASSERT(hipMalloc(&image_D, n_bytes));
  
  /*Copy the input image to the buffer */
	CUDA_ASSERT(hipMemcpy(image_D, source->data, n_bytes, hipMemcpyHostToDevice));
	
  /*Calculate the block size and the number of thread blocks */
	const dim3 blocksize(16, 16);
  const dim3 numblocks((source->width % blocksize.x) ?
                          source->width / blocksize.x + 1 :
                          source->width / blocksize.x,
                       (source->height % blocksize.y) ?
                          source->height / blocksize.y + 1 :
                          source->height / blocksize.y);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  for (int Z = 0; Z < n_repeat; Z++)
    {
      op_cuda_selgray<<<numblocks, blocksize>>>(image_D, 
      																					source->rowstride, 
      																					0, 0,
      																					source->width,
      																					source->height,
      																					40, 30);
      
      CUDA_ASSERT(hipGetLastError());
    }

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));
	
	CUDA_ASSERT(hipMemcpy(source->data, image_D, n_bytes, hipMemcpyDeviceToHost));
	
	CUDA_ASSERT(hipFree(image_D));
	
  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));
  fprintf(stderr, "elapsed time GPU selgray: %f s\n", msec / 1000.);
}


image_t *
run_op_light2d(const image_t *source, const int n_repeat)
{
#if 0
  /* TODO: enable once you need this. */
  /* Define two example light sources */
  static const light_t lights[] =
    {
      {
        0.75,      // X
        0.75,      // Y
        2.0,       // Height above image, to produce faux Z-axis
        { 0.7, 0.5, 0.5, 1.0 } // Intensity RGBA
      },
      {
        0.25,      // X
        0.25,      // Y
        2.0,       // Height
        { 0.5, 0.7, 0.7, 1.0 } // Intensity RGBA
      }
    };

  const int n_lights = sizeof(lights) / sizeof(lights[0]);
#endif

  /* TODO: Allocate buffers to contain initial image and result */

  /* TODO: Copy the input image to the initial buffer */

  /* TODO: Calculate the block size and the number of thread blocks */


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  for (int Z = 0; Z < n_repeat; Z++)
    {
      /* TODO: call CUDA kernel */
    }

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  image_t *result = image_new_from_image(source);
  /* TODO: copy back result */

  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));
  fprintf(stderr, "elapsed time GPU light2d: %f s\n", msec / 1000.);

  return result;
}


} /* end extern "C" */
